#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
// #include /usr/local/cuda/include/hip/hip_runtime.h

//add vector kernel
__global__ void addVector(int *a,int *b,int*c,int n){

  //used to calcute unique thread index, which represents position of thread on entire grid
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
        c[i] =a[i]+b[i];
    }   
}

//matrix multiplication kernel
__global__ void matrixMultiplication(int *a,int *b, int *c , int size){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<size && col<<size){
        int product_sum=0;
        for(int k=0;k<size;k++){
            product_value += A[row*size + k] * B[k*size +col]
        }
        c[row*n+col] = product_value[];
    }
}

//utility functions
void initializeVector(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void initializeMatrix(int* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = rand() % 10;
    }
}

void printVector(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", vector[i]);
    }
    printf("\n");
}

void printMatrix(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            printf("%d ", matrix[row * size + col]);
        }
        printf("\n");
    }
    printf("\n");
}



int main(){

int n=4;

//===============vector addition=================

int *A,*B,*C;
int size=n*sizeof(int);

//allocate memory on host
hipHostMalloc(&A,size);
hipHostMalloc(&B,size);
cudaMAllocHost(&C,size);

//initialize the vectors
initializeVector(A,n);
initializeVector(B,n);

printf("Vector A: \n");
printVector(A, size);
printf("Vector B: \n");
printVector(B, size);

//allocate memory on device
int *device_A,*device_B,*device_C;
hipMalloc(&device_A,size);
hipMalloc(&device_B,size);
hipMalloc(&device_C,size);

//copy data from host to device
hipMemcpy(device_A,A,size,hipMemcpyHostToDevice);
hipMemcpy(device_B,B,size,hipMemcpyHostToDevice);

//launch cuda kernel
int threadsPerBlockVec = 256;
int blocksPerGridVec = (n+threadsPerBlockVec-1)/threadsPerBlockVec;
add<<<blocksPerGridVec,threadsPerBlockVec>>> addVector(device_A,device_B,device_C);

//copy data from device to host 
hipMemcpy(C,dev_c,size,hipMemcpyDeviceToHost);

//print ouput/result
printVector(c,n);

//free previously allocated device memory 
hipFree(device_A);
hipFree(device_B);
hipFree(device_C);

//free previously allocated host memory 
hipHostFree(A);
hipHostFree(B);
hipHostFree(C);


//=============Matrix multiplication================

int *D, *E,*F;
size_t matrix_size = n*n*sizeof(int);

//allocate memory to host
cudaMAllocHost(&D,matrix_size);
hipHostMalloc(&E,matrix_size);
hipHostMalloc(&F,matrix_size);

initializeMatrix(D, matrixSize);
initializeMatrix(E, matrixSize);

printf("\nMatrix D: \n");
printMatrix(D, matrixSize);

printf("Matrix E: \n");
printMatrix(E, matrixSize);

int *d_D,*d_E,*d_F;

//allocate device memory to D,E and F matrix
hipMalloc(&d_D,size);
hipMalloc(&d_E,size);
cudaMAlloc(&d_F,size);


hipMemcpy(d_D, D, matrix_size, hipMemcpyHostToDevice);
hipMemcpy(d_E, E, matrix_size, hipMemcpyHostToDevice);

int threadsPerBlockMat = 2;
int blocksPerGridMat = (matrix_size + threadsPerBlockMat - 1) / threadsPerBlockMat;

dim3 threadsMat(threadsPerBlockMat, threadsPerBlockMat);
dim3 blocksMat(blocksPerGridMat, blocksPerGridMat);

multiply<<<blocksMat, threadsMat>>>(d_D, d_E, d_F, matrix_size);

hipMemcpy(F, s_F, matrix_size, hipMemcpyDeviceToHost);
printf("Multiplication: \n");
printMatrix(F, matrix_size);

free(D);
free(E);
free(F);

hipFree(d_D);
hipFree(d_E);
hipFree(d_F);

    return 0;
}


// nvcc programName.cu - o programName
// ./programName or programName
