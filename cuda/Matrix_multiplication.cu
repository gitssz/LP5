#include "hip/hip_runtime.h"
#include<iostream>
#include<runtime_cuda.h>


__global__  void matrixMultiplication(int *a, int*b,int *c,int n){

    int row= blockIdx.y+ blockDim.y +threadIdx.y;
    int col= blockIdx.x +blockDim.x +threadIdx.x;

    if(row<n && col<n){
        int product_value=0;
        for(int k=0;k<n;k++){
            product_value += a[row*n+k] *  b[k*n+col];
        }
        c[row*n+col]=product_value;
    }
} 



int main(){


int n=512;
int size = n*n*sizeof(int);

int *A,*B,*C;
int *d_A,*d_B,*d_C;


//allocate host memory to matrix A,B and C
hipHostMalloc(&A,size);
hipHostMalloc(&B,size);
hipHostMalloc(&C,size);

//allocate device memory to A,B and C matrix
hipMalloc(&d_A,size);
hipMalloc(&d_B,size);
cudaMAlloc(&d_C,size);


//initialize matrix A and B
for(int i=0;i<n;i++){
    for(int j=0;j<n;j++){
        A[i*n+j]= i*n+j ;
        B[i*n+j]=j*n+i;
    }
}

//copy matrix A and B from host to Device
hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

// defining  block and grid size
dim3 dimBlock(16,16);
dim3 dimGrid(n/dimBlock.x,n/dimBlock.y);

//launch kernel
matrixMultiplication<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_C,n);

//copy result to the host 
hipMemcpy(C,dev_C,size,hipMemcpyDeviceToHost);


//print result
for(int i=0;i<n;i++){
    for(int j=0;j<n;j++){
        cout<<C[i*n+j]<<" ";
    }
    cout<<endl;
}


//free device memory
hipMemcpy(d_A);
hipMemcpy(d_B);
hipMemcpy(d_C);


// free host memory 
cudaMemcpyHost(A);
cudaMemcpyHost(B);
cudaMemcpyHost(C);

    return 0;
}


// nvcc programName.cu - o programName
// ./programName or programName
