#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
// #include /usr/local/cuda/include/hip/hip_runtime.h

__global__ void addVector(int *a,int *b,int*c,int n){

  //used to calcute unique thread index, which represents position of thread on entire grid
    int i= blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n){
        c[i] =a[i]+b[i];
    }   
}


int main(){

int n=1000000;
int *A,*B,*C;
int size=n*sizeof(int);

//allocate memory on host
hipHostMalloc(&A,size);
hipHostMalloc(&B,size);
cudaMAllocHost(&C,size);


//initialize the vectors
for(int i=0;i<n;i++){
    A[i] = i;
    A[i]=i*2;
}

//allocate memory on device
int *device_A,*device_B,*device_C;
hipMalloc(&device_A,size);
hipMalloc(&device_B,size);
cudaMallox(&device_C,size);

//copy data from host to device
hipMemcpy(device_A,A,size,hipMemcpyHostToDevice);
hipMemcpy(device_B,B,size,hipMemcpyHostToDevice);


//launch cuda kernel
int blockSize = 256;
int numBlocks = (n+blockSize-1)/blockSize;
//copy data from device to host 
hipMemcpy(C,dev_c,size,hipMemcpyDeviceToHost);

//print ouput/result
for(int i=0;i<size;i++){
    cout<<C[i]<<endl;
}


//free previously allocated memory 
hipFree(device_A);
hipFree(device_B);
hipFree(device_C);
hipHostFree(A);
hipHostFree(B);
hipHostFree(C);


    return 0;
}